
#include <hip/hip_runtime.h>

#define conductivity 0.1666666f

// Convert an xyz coordinate to an index in the array
__device__ __forceinline__ int toIndex(int x, int y, int z, int cellsWide) {
	//return x * cellsWide * cellsWide + y * cellsWide + z;
	return (x * cellsWide + y) * cellsWide + z;
}

// Compute one step of thermal diffusion in 3D
extern "C"
__global__ void thermalDiffusionStep(const int cellsWide,
		const int airPadding, const char shouldComputeAir, const char shouldComputeBoundary,
		const char* cellEnabled, float* cellHeat, float* cellDeltaHeat, const char copyMode,
		const float heatSourceHeatPerCell) {
	
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int z = blockIdx.z * blockDim.z + threadIdx.z;
	
	if (x >= cellsWide || y >= cellsWide || z >= cellsWide) {
		return;
	}
	
	const int idx = toIndex(x, y, z, cellsWide);
	
	// If we only want to copy the results from the previous iteration
	if (copyMode) {
		cellHeat[idx] += cellDeltaHeat[idx];
	} else {
		// Array to represent the 6 adjacent cells
		const char axes[] = {
				1, 0, 0,
				-1, 0, 0,
				0, 1, 0,
				0, -1, 0,
				0, 0, 1,
				0, 0, -1};
		
		// If this cell gets heat directly from the artificial heat source
		if (y == 0 && x > airPadding*1.3f-1 && x < cellsWide-airPadding*1.3f &&
					z > airPadding*1.3f-1 && z < cellsWide-airPadding*1.3f) {
			cellDeltaHeat[idx] = heatSourceHeatPerCell;
		} else {
			cellDeltaHeat[idx] = 0;
		}
		
		// Iterate over the adjacent cells
		for (int i = 0; i < 6 * 3; i += 3) {
			// Check if this cell is in bounds
			if (x + axes[i+0] >= 0 && x + axes[i+0] < cellsWide &&
					y + axes[i+1] >= 0 && y + axes[i+1] < cellsWide &&
					z + axes[i+2] >= 0 && z + axes[i+2] < cellsWide) {
				
				const int idx2 = toIndex(x + axes[i+0], y + axes[i+1], z + axes[i+2], cellsWide);
				
				if (cellEnabled[idx] != cellEnabled[idx2]) {
					// Air to metal boundary (exclude the heat source area)
					if (!shouldComputeBoundary) {
						continue;
					}
				} else if (cellEnabled[idx] == 0) {
					// Air to air boundary
					if (!shouldComputeAir) {
						continue;
					}
				} // Metal to metal boundary is always computed
				
				cellDeltaHeat[idx] += (cellHeat[idx2] - cellHeat[idx]) * conductivity;
			} else if (shouldComputeAir && !(y == 0 &&
						x > airPadding*1.3f-1 && x < cellsWide-airPadding*1.3f &&
						z > airPadding*1.3f-1 && z < cellsWide-airPadding*1.3f)) {
				// This cell is not in bounds, so apply border conditions.
				// The border heat is assumed to be ambient temperature air (0 degrees).
				cellDeltaHeat[idx] -= cellHeat[idx] * conductivity;
			}
		}
	}
}